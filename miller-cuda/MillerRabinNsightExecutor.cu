#include "hip/hip_runtime.h"
#include "Utils.h"
#include "MillerRabinNsightExecutor.cuh"

__device__ uint64_t cudaSpecificRandomNsight(hiprandState* state, uint64_t min, uint64_t max) {

    // Zwrócenie min gdy zakres jest niewłaściwy
    if (min > max) {
        return min;
    }
    return hiprand(state) % (max - min + 1) + min;
}

__global__ void miller_rabin_nsight_kernel(uint64_t* number, int iterations, bool* result, hiprandState* states) {
    // Pamięć współdzielona
    __shared__ bool shared_results[256];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= iterations) return;

    // Wywołanie hiprand_init raz na wątek zamiast dla każdego wątku
    //hiprandState localState;
    //hiprand_init(clock64(), idx, 0, &localState);

    //const uint64_t base = cudaSpecificRandomNsight(&states[idx], 2, *number - 2);
    //uint64_t exponent_of_two, odd_part;
    //Utils::decompose_number(*number - 1, exponent_of_two, odd_part);

    // Inicjalizacja stanu generatora liczb losowych
    hiprandState localState = states[idx];

    const uint64_t base = cudaSpecificRandomNsight(&localState, 2, *number - 2);
    uint64_t exponent_of_two, odd_part;
    Utils::decompose_number(*number - 1, exponent_of_two, odd_part);

    uint64_t x = Utils::modular_pow(base, odd_part, *number);

    if (x == 1 || x == *number - 1) {
        shared_results[threadIdx.x] = true;
        return;
    }

    for (uint64_t j = 0; j < exponent_of_two - 1; ++j) {
        x = Utils::modular_pow(x, 2, *number);
        if (x == *number - 1) {
            shared_results[threadIdx.x] = true;
            return;
        }
    }

    shared_results[threadIdx.x] = false;

    // Synchronizacja wątków w bloku przed zbieraniem wyników
    __syncthreads();

    // Agregowanie wyników w obrębie bloku
    if (threadIdx.x == 0) {
        bool all_true = true;
        for (int i = 0; i < blockDim.x; ++i) {
            if (!shared_results[i]) {
                all_true = false;
                break;
            }
        }
        result[blockIdx.x] = all_true;
    }
}

bool miller_rabin_test_gpu_nsight(uint64_t number, int iterations) {

    // Szybka obsługa małych liczb
    if (number <= 3) {
        return number > 1;
    }

    uint64_t* d_number;
    bool* d_results, * h_results;
    h_results = new bool[iterations];

    // Alokacja pami�ci na GPU
    hipMalloc((void**)&d_number, sizeof(uint64_t));
    hipMalloc((void**)&d_results, iterations * sizeof(bool));
    hiprandState* d_states;
    hipMalloc(&d_states, iterations * sizeof(hiprandState));

    // Przekazywanie liczby do GPU
    hipMemcpy(d_number, &number, sizeof(uint64_t), hipMemcpyHostToDevice);

    // Ustalanie liczby blok�w i w�tk�w
    int threads_per_block = 256;
    int blocks_per_grid = (iterations + threads_per_block - 1) / threads_per_block;
    miller_rabin_nsight_kernel << <blocks_per_grid, threads_per_block >> > (d_number, iterations, d_results, d_states);

    // Kopiowanie wynik�w z GPU
    hipMemcpy(h_results, d_results, iterations * sizeof(bool), hipMemcpyDeviceToHost);

    bool is_prime = true;
    for (int i = 0; i < iterations; ++i) {
        if (!h_results[i]) {
            is_prime = false;
            break;
        }
    }

    // Zwolnienie pami�ci
    hipFree(d_number);
    hipFree(d_results);
    hipFree(d_states);
    delete[] h_results;

    return is_prime;
}